//
// Threshold-based Contrasting GPU implementation
//

#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdlib.h>

#define THREADS_PER_BLOCK		128

// CUDA kernel declaration
__global__ void cuda_flow_bitmap_kernel(unsigned char *u8data, unsigned char *u8res, unsigned int N, unsigned int thresh);

// C/C++ Wrapper
unsigned char *gpu_flow_recogu8_bitmap(unsigned char *u8data, unsigned char *u8res, unsigned int N, unsigned int thresh)
{
	// Host memory
	unsigned char *host_u8res;

	// Device memory
	unsigned char *dev_u8data;
	unsigned char *dev_u8res;

	size_t size = N * sizeof(unsigned char);
	host_u8res = (unsigned char *)malloc(size);

	// Allocated device memory
	hipMalloc((void **)&dev_u8data, size);
	hipMalloc((void **)&dev_u8res, size);

	// Upload data to device memory
	hipMemcpy(dev_u8data, u8data, size, hipMemcpyHostToDevice);

	cuda_flow_bitmap_kernel<<<((N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK), THREADS_PER_BLOCK>>>(dev_u8data, dev_u8res, N, thresh);

	hipMemcpy(host_u8res, dev_u8res, size, hipMemcpyDeviceToHost);

	hipFree(dev_u8data);
	hipFree(dev_u8res);

	return host_u8res;
}

// u8res contains a binary map of optical flow regions with motion intensity over the threshold
__global__ void cuda_flow_bitmap_kernel(unsigned char *u8data, unsigned char *u8res, unsigned int N, unsigned int thresh)
{
	int idx = threadIdx.x + (blockDim.x *blockIdx.x);

	if (idx < N) {

		if (u8data[idx] >= thresh) {
			u8res[idx] = 1;
		}
		else {
			u8res[idx] = 0;
		}
	}
}



