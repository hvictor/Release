/*
 * gpu_physical_model.c
 *
 *  Created on: Jan 24, 2016
 *      Author: sled
 */

//
// Physical model GPU implementation
//

#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>

#define THREADS_PER_BLOCK		128

// CUDA kernel declaration
__global__ void cuda_physical_model_kernel(double g, double K, double vx_0, double vy_0, double vz_0, double *x, double *y, double *z, int N);

// C/C++ Wrapper
void gpu_physical_model_compute(double g, double K, double vx_0, double vy_0, double vz_0, double *x, double *y, double *z, int N)
{
	// Device memory
	double *dev_x;
	double *dev_y;
	double *dev_z;

	size_t size = N * sizeof(double);

	// Allocated device memory
	hipMalloc((void **)&dev_x, size);
	hipMalloc((void **)&dev_y, size);
	hipMalloc((void **)&dev_z, size);

	// Upload data to device memory
	hipMemcpy(dev_x, x, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_y, y, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_z, z, size, hipMemcpyHostToDevice);

	cuda_physical_model_kernel<<<((N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK), THREADS_PER_BLOCK>>>(g, K, vx_0, vy_0, vz_0, dev_x, dev_y, dev_z, N);

	hipMemcpy(x, dev_x, size, hipMemcpyDeviceToHost);
	hipMemcpy(y, dev_y, size, hipMemcpyDeviceToHost);
	hipMemcpy(z, dev_z, size, hipMemcpyDeviceToHost);

	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_z);
}

// GPU memory buffers are filled with computed physical states 3D components
__global__ void cuda_physical_model_kernel(double g, double K, double vx_0, double vy_0, double vz_0, double *x, double *y, double *z, int N)
{
	int idx = threadIdx.x + (blockDim.x *blockIdx.x);

	if (idx == 0)
		return;

	if (idx < N) {
		x[idx] = x[0] + (vx_0 / K) * (1.0 - exp(-K * idx)/K);
		y[idx] = y[0] + ((g + K * vy_0) / (K * K)) * (1.0 - exp(-K * idx)) - (g * idx) / K;
		z[idx] = z[0] + (vz_0 / K) * (1.0 - exp(-K * idx)/K);
	}
}



