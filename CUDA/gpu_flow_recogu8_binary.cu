/*
 * cuda_flow_recog8u_binary.c
 *
 *  Created on: Jan 27, 2016
 *      Author: sled
 */

//
// Threshold-based Contrasting GPU implementation
//

#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>

#define THREADS_PER_BLOCK		128

// CUDA kernel declaration
__global__ void cuda_flow_binary_kernel(float *flowx, float *flowy, unsigned char *u8res, float thresh, int N);

// C/C++ Wrapper
void gpu_flow_recogu8_binary(float *flowx, float *flowy, unsigned char *u8res, float thresh, int N)
{
	// Device memory
	float *dev_flowx;
	float *dev_flowy;
	uint8_t *dev_u8res;

	// Allocated device memory
	hipMalloc((void **)&dev_flowx, N * sizeof(float));
	hipMalloc((void **)&dev_flowy, N * sizeof(float));
	hipMalloc((void **)&dev_u8res, N * sizeof(uint8_t));

	// Upload data to device memory
	hipMemcpy(dev_flowx, flowx, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_flowy, flowy, N * sizeof(float), hipMemcpyHostToDevice);

	cuda_flow_binary_kernel<<<((N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK), THREADS_PER_BLOCK>>>(dev_flowx, dev_flowy, dev_u8res, thresh, N);

	hipMemcpy(u8res, dev_u8res, N * sizeof(uint8_t), hipMemcpyDeviceToHost);

	hipFree(dev_flowx);
	hipFree(dev_flowy);
}

// u8res contains a binary map of optical flow regions with motion intensity over the threshold
__global__ void cuda_flow_binary_kernel(float *flowx, float *flowy, unsigned char *u8res, float thresh, int N)
{
	int idx = threadIdx.x + (blockDim.x *blockIdx.x);

	if (idx < N) {

		if (isnan(flowx[idx]) || isnan(flowy[idx]) || fabs(flowx[idx]) < 1e9 || fabs(flowy[idx]) < 1e9)
			return;

		if (sqrt(flowx[idx]*flowx[idx] + flowy[idx]*flowy[idx]) >= thresh) {
			u8res[idx] = 1;
		}
		else {
			u8res[idx] = 0;
		}
	}
}






