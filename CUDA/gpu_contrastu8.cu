//
// Threshold-based Contrasting GPU implementation
//

#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdlib.h>

#define THREADS_PER_BLOCK		128

// CUDA kernel declaration
__global__ void cuda_contrast_kernel(unsigned char *u8data, unsigned char *u8res, unsigned int N, unsigned int thresh, unsigned int delta);

// C/C++ Wrapper
unsigned char *gpu_contrastu8(unsigned char *u8data, unsigned char *u8res, unsigned int N, unsigned int thresh, unsigned int delta)
{
	// Host memory
	unsigned char *host_u8res;

	// Device memory
	unsigned char *dev_u8data;
	unsigned char *dev_u8res;

	size_t size = N * sizeof(unsigned char);
	host_u8res = (unsigned char *)malloc(size);

	// Allocated device memory
	hipMalloc((void **)&dev_u8data, size);
	hipMalloc((void **)&dev_u8res, size);

	// Upload data to device memory
	hipMemcpy(dev_u8data, u8data, size, hipMemcpyHostToDevice);

	cuda_contrast_kernel<<<((N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK), THREADS_PER_BLOCK>>>(dev_u8data, dev_u8res, N, thresh, delta);

	hipMemcpy(host_u8res, dev_u8res, size, hipMemcpyDeviceToHost);

	hipFree(dev_u8data);
	hipFree(dev_u8res);

	return host_u8res;
}

// CUDA kernel
__global__ void cuda_contrast_kernel(unsigned char *u8data, unsigned char *u8res, unsigned int N, unsigned int thresh, unsigned int delta)
{
	int idx = threadIdx.x + (blockDim.x *blockIdx.x);

	if (idx < N) {

		if (u8data[idx] >= thresh) {
			if (u8data[idx] + delta >= 255) {
				u8res[idx] = 255;
			}
			else {
				u8res[idx] = u8data[idx] + delta;
			}
		}
		else {
			if (u8data[idx] - delta <= 0) {
				u8res[idx] = 0;
			}
			else {
				u8res[idx] = u8data[idx] - delta;
			}
		}
	}
}
