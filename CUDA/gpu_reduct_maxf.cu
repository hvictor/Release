#include "hip/hip_runtime.h"
/*
 * gpu_maxf.c
 *
 *  Created on: Jan 24, 2016
 *      Author: sled
 */

__device__ float atomicMaxf(float* address, float val)
{
	int *address_as_int =(int*)address;
	int old = *address_as_int, assumed;
	while (val > __int_as_float(old)) {
		assumed = old;
		old = atomicCAS(address_as_int, assumed,
				__float_as_int(val));
	}
	return __int_as_float(old);
}


__global__ void cuda_reduct_maxf_kernel(const float* const d_array, float* d_max,  const size_t elements)
{
	extern __shared__ float shared[];

	int tid = threadIdx.x;
	int gid = (blockDim.x * blockIdx.x) + tid;
	shared[tid] = -FLOAT_MAX;

	while (gid < elements) {
		shared[tid] = max(shared[tid], d_array[gid]);
		gid += gridDim.x*blockDim.x;
	}
	__syncthreads();
	gid = (blockDim.x * blockIdx.x) + tid;  // 1
	for (unsigned int s=blockDim.x/2; s>0; s>>=1)
	{
		if (tid < s && gid < elements)
			shared[tid] = max(shared[tid], shared[tid + s]);
		__syncthreads();
	}

	if (tid == 0)
		atomicMaxf(d_max, shared[0]);
}


